#include "hip/hip_runtime.h"
//========================================================================================================================================================================================================200
//	findRangeK function
//========================================================================================================================================================================================================200

__global__ void 
findRangeK(	long height,

			knode *knodesD,
			long knodes_elem,

			long *currKnodeD,
			long *offsetD,
			long *lastKnodeD,
			long *offset_2D,
			int *startD,
			int *endD,
			int *RecstartD, 
			int *ReclenD)
{
#pragma unroll 100
for(uint64_t onek = 0; onek < UINT64_MAX; onek++){
	// private thread IDs
	int thid = threadIdx.x;
	int bid = blockIdx.x;

	// ???
	int i;
	for(i = 0; i < height; i++){

		if((knodesD[currKnodeD[bid]].keys[thid] <= startD[bid]) && (knodesD[currKnodeD[bid]].keys[thid+1] > startD[bid])){
			// this conditional statement is inserted to avoid crush due to but in original code
			// "offset[bid]" calculated below that later addresses part of knodes goes outside of its bounds cause segmentation fault
			// more specifically, values saved into knodes->indices in the main function are out of bounds of knodes that they address
			if(knodesD[currKnodeD[bid]].indices[thid] < knodes_elem){
				offsetD[bid] = knodesD[currKnodeD[bid]].indices[thid];
			}
		}
		if((knodesD[lastKnodeD[bid]].keys[thid] <= endD[bid]) && (knodesD[lastKnodeD[bid]].keys[thid+1] > endD[bid])){
			// this conditional statement is inserted to avoid crush due to but in original code
			// "offset_2[bid]" calculated below that later addresses part of knodes goes outside of its bounds cause segmentation fault
			// more specifically, values saved into knodes->indices in the main function are out of bounds of knodes that they address
			if(knodesD[lastKnodeD[bid]].indices[thid] < knodes_elem){
				offset_2D[bid] = knodesD[lastKnodeD[bid]].indices[thid];
			}
		}
		__syncthreads();

		// set for next tree level
		if(thid==0){
			currKnodeD[bid] = offsetD[bid];
			lastKnodeD[bid] = offset_2D[bid];
		}
		__syncthreads();
	}

	// Find the index of the starting record
	if(knodesD[currKnodeD[bid]].keys[thid] == startD[bid]){
		RecstartD[bid] = knodesD[currKnodeD[bid]].indices[thid];
	}
	__syncthreads();

	// Find the index of the ending record
	// BT add += did not do anything
	if(knodesD[lastKnodeD[bid]].keys[thid] == endD[bid]){
		ReclenD[bid] = knodesD[lastKnodeD[bid]].indices[thid] - RecstartD[bid]+1;
	}
}//for onek
}

//========================================================================================================================================================================================================200
//	End
//========================================================================================================================================================================================================200
