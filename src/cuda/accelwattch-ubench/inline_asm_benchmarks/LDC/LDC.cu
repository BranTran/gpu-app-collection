#include "hip/hip_runtime.h"
// Copyright (c) 2018-2021, Vijay Kandiah, Junrui Pan, Mahmoud Khairy, Scott Peverelle, Timothy Rogers, Tor M. Aamodt, Nikos Hardavellas
// Northwestern University, Purdue University, The University of British Columbia
// All rights reserved.
//
// Redistribution and use in source and binary forms, with or without
// modification, are permitted provided that the following conditions are met:
//
// 1. Redistributions of source code must retain the above copyright notice, this
//    list of conditions and the following disclaimer;
// 2. Redistributions in binary form must reproduce the above copyright notice,
//    this list of conditions and the following disclaimer in the documentation
//    and/or other materials provided with the distribution;
// 3. Neither the names of Northwestern University, Purdue University,
//    The University of British Columbia nor the names of their contributors
//    may be used to endorse or promote products derived from this software
//    without specific prior written permission.
//
// THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
// AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
// IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE
// ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE
// LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR
// CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF
// SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS
// INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN
// CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE)
// ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE
// POSSIBILITY OF SUCH DAMAGE.
// Includes
#include <stdio.h>
#include <stdlib.h>


// includes CUDA
#include <hip/hip_runtime.h>


#define THREADS_PER_BLOCK 256
#define NUM_OF_BLOCKS 640

__constant__ unsigned ConstArray1[THREADS_PER_BLOCK];



unsigned* h_Value;
unsigned* d_Value;


// Functions
void CleanupResources(void);
void RandomInit(unsigned*, int);
FILE *fp;


////////////////////////////////////////////////////////////////////////////////
// These are CUDA Helper functions

// This will output the proper CUDA error strings in the event that a CUDA host call returns an error
#define checkCudaErrors(err)  __checkCudaErrors (err, __FILE__, __LINE__)

inline void __checkCudaErrors(hipError_t err, const char *file, const int line )
{
  if(hipSuccess != err){
	fprintf(stderr, "%s(%i) : CUDA Runtime API error %d: %s.\n",file, line, (int)err, hipGetErrorString( err ) );
	 exit(-1);
  }
}

// This will output the proper error string when calling hipGetLastError
#define getLastCudaError(msg)      __getLastCudaError (msg, __FILE__, __LINE__)

inline void __getLastCudaError(const char *errorMessage, const char *file, const int line )
{
  hipError_t err = hipGetLastError();
  if (hipSuccess != err){
	fprintf(stderr, "%s(%i) : getLastCudaError() CUDA error : %s : (%d) %s.\n",file, line, errorMessage, (int)err, hipGetErrorString( err ) );
	exit(-1);
  }
}

// end of CUDA Helper Functions


__global__ void PowerKernal(unsigned* Value, unsigned long long iterations)
{
	int tid = threadIdx.x;
	int i = blockIdx.x*THREADS_PER_BLOCK + tid;

	unsigned Value1=0;
    unsigned sink=0;
	#pragma unroll 100
    for(unsigned long long k=0; k<iterations;k++) {
		asm volatile(
			"ld.const.u32 %0, [%1];\t"
			: "=r"(Value1)
			: "l"(ConstArray1)
		);
        sink += Value1;
	}
		Value[i] = sink;
}



// Host code

int main(int argc, char** argv) 
{
    unsigned long long iterations;
    if (argc != 2){
        fprintf(stderr,"usage: %s #iterations\n",argv[0]);
        exit(1);
    }
    else{
        iterations = atoll(argv[1]);
    }

    printf("Power Microbenchmark with %d iterations\n",iterations);
	 unsigned array1[THREADS_PER_BLOCK];
	int N = THREADS_PER_BLOCK*NUM_OF_BLOCKS;
	size_t size = N * sizeof(unsigned);
	 h_Value = (unsigned *) malloc(size);

	// Initialize input vectors
	RandomInit(array1, THREADS_PER_BLOCK);

	 hipMemcpyToSymbol(HIP_SYMBOL(ConstArray1), array1, sizeof(unsigned) * THREADS_PER_BLOCK );
	 
	 checkCudaErrors( hipMalloc((void**)&d_Value, size ));
	 //VecAdd<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, N);
	 dim3 dimGrid(NUM_OF_BLOCKS,1);
	 dim3 dimBlock(THREADS_PER_BLOCK,1);

	 hipEvent_t start, stop;
	  float elapsedTime = 0;
	  checkCudaErrors(hipEventCreate(&start));
	  checkCudaErrors(hipEventCreate(&stop));

	  checkCudaErrors(hipEventRecord(start));
	  PowerKernal<<<dimGrid,dimBlock>>>(d_Value, iterations);
	  checkCudaErrors(hipEventRecord(stop));
	 
	  checkCudaErrors(hipEventSynchronize(stop));
	  checkCudaErrors(hipEventElapsedTime(&elapsedTime, start, stop));
	  printf("gpu execution time = %.3f ms\n", elapsedTime);
	  getLastCudaError("kernel launch failure");
	  checkCudaErrors( hipMemcpy(h_Value, d_Value, size, hipMemcpyDeviceToHost) );
 	  checkCudaErrors(hipEventDestroy(start));
	  checkCudaErrors(hipEventDestroy(stop));
	  return 0;
}

// Allocates an array with random float entries.
void RandomInit(unsigned* data, int n)
{
  for (int i = 0; i < n; ++i){
  srand((unsigned)time(0));  
  data[i] = rand() / RAND_MAX;
  }
}





